
#include <hip/hip_runtime.h>
extern "C" __global__ __launch_bounds__(32) void sconv_bprop_C1_N64(
  float *param_test,
  float *param_I,
  const float *param_E,
  const float *param_F,
  float param_alpha,   
  float param_N,
  float param_K,            
  float param_D,            
  float param_H,            
  float param_W,            
  float param_WN,          
  float param_HWN,          
  float param_DHWN,         
  float param_C,            
  float param_CRST,         
  float param_RST,          
  float param_magic_RST,    
  float param_shift_RST,    
  float param_RS,           
  float param_magic_RS,     
  float param_shift_RS,     
  float param_S,            
  float param_magic_S,      
  float param_shift_S,      
  float param_pad_d,        
  float param_pad_h,        
  float param_pad_w,        
  float param_str_d,        
  float param_str_h,        
  float param_str_w,        
  float param_Q,            
  float param_PQ,           
  float param_QN,           
  float param_PQN,          
  float param_MPQN,         
  float param_magic_Q,      
  float param_shift_Q,      
  float param_magic_PQ,     
  float param_shift_PQ,     
  float param_CRST8,        
  float param_MPQN8) {
  __shared__ float share[64 * 8 * 2 + 32 * 8 * 2 + 8];
  int tid = threadIdx.x;
  share[tid] = 1;
}